#include <thread>
#include <iostream>
#include <vector>
#include <algorithm>
#include <chrono>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>  
#include <hipblas.h>
#include <hipblaslt.h>
#include "../third_party/cuda/gpu.h"
using namespace std::chrono;
const int N = 20000;

void set_to_one(float *blk) {
  for (size_t idx = 0; idx < N * N; ++idx) {
    blk[idx] = 1.0f;
  }
}

int main() {

  float *a_blk = (float *)malloc(sizeof(float) * N * N);
  set_to_one(a_blk);
  float *b_blk = (float *)malloc(sizeof(float) * N * N);
  set_to_one(b_blk);
  float *c_blk = (float *)malloc(sizeof(float) * N * N);

  auto start = high_resolution_clock::now();
  hipblasHandle_t cublas_handle;
  hipblasCreate(&cublas_handle);
  
  float *a_gpu_blk, *b_gpu_blk, *c_gpu_blk;
  checkCudaErrors(hipMalloc(&a_gpu_blk, N * N * sizeof(float)));
  checkCudaErrors(hipMalloc(&b_gpu_blk, N * N * sizeof(float)));
  checkCudaErrors(hipMalloc(&c_gpu_blk, N * N * sizeof(float)));
  hipMemcpy(a_gpu_blk, a_blk, N * N * sizeof(float), hipMemcpyDeviceToDevice);
  hipMemcpy(b_gpu_blk, b_blk, N * N * sizeof(float), hipMemcpyDeviceToDevice);
  hipDeviceSynchronize();
  auto stop = high_resolution_clock::now();
  auto duration = duration_cast<microseconds>(stop - start);
  std::cout << "Time it took : " << (float)duration.count() * 1e-6f << std::endl;

  start = high_resolution_clock::now();
  float alpha = 1.0f;
  float beta = 0.0f;
  hipblasSgemm(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha, a_gpu_blk, N, b_gpu_blk, N, &beta, c_gpu_blk, N);
  hipDeviceSynchronize();
  stop = high_resolution_clock::now();
  duration = duration_cast<microseconds>(stop - start);
  std::cout << "Time it took for the kernel: " << (float)duration.count() * 1e-6f << std::endl;

  return 0;
}