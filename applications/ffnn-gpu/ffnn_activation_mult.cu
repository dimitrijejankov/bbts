#include "hip/hip_runtime.h"
#include "ffnn_activation_mult.h"
#include "ffnn_types.h"
#include <cassert>
#include <hipblas.h>

bbts::ffnn_activation_mult::ffnn_activation_mult() {

  // set the names
  impl_name = "ffnn_act_mult_gpu";
  ud_name = "ffnn_act_mult";

  // set the input and output types
  inputTypes = {"ffnn_dense", "ffnn_dense"};
  outputTypes = {"ffnn_dense"};

  // both inputs zero and one can be used as the inplace output
  inputInplace = {};

  // this is a CPU dense mult
  is_gpu = true;

  // set the function that actually performs the add
  fn = &ffnn_activation_mult::mult;
}

size_t bbts::ffnn_activation_mult::get_complexity_hint(
    const bbts::ud_impl_t::tensor_params_t &params,
    const bbts::ud_impl_t::meta_args_t &_in) {

  // O(n * m * k)
  const auto &m_a = _in.get<0>().as<ffnn_dense_meta_t>().m();
  const auto &m_b = _in.get<1>().as<ffnn_dense_meta_t>().m();
  return 1.45838e-11 * m_a.num_rows * m_a.num_cols * m_b.num_cols;
}

__global__ void ffnn_activation_mult_bias_add_kernel(float *b, float *c,
                                                     int num_rows,
                                                     int num_cols) {

  // get our global thread ID
  int row = blockIdx.x * blockDim.x + threadIdx.x;
  int col = blockIdx.y * blockDim.y + threadIdx.y;

  // make sure we do not go out of bounds
  if (row < num_rows && col < num_cols)
    c[col * num_rows + row] += b[col];
}

void bbts::ffnn_activation_mult::get_out_meta(
    const bbts::ud_impl_t::tensor_params_t &params,
    const bbts::ud_impl_t::meta_args_t &_in,
    bbts::ud_impl_t::meta_args_t &_out) const {

  // get the input argeters
  const auto &m_a = _in.get<0>().as<ffnn_dense_meta_t>().m();
  const auto &m_b = _in.get<1>().as<ffnn_dense_meta_t>().m();

  // get the output argeters
  auto &m_out = _out.get<0>().as<ffnn_dense_meta_t>().m();

  // set the output
  uint32_t I = m_a.num_rows;
  uint32_t J = m_b.num_cols;
  m_out = {.num_rows = I,
           .num_cols = J,
           .row_idx = m_a.row_idx,
           .col_idx = m_b.col_idx,
           .has_bias = false,
           .num_aggregated = 1};

  auto num_elements = m_out.num_cols * m_out.num_rows;
  num_elements += m_out.has_bias ? m_out.num_cols : 0;
}

void bbts::ffnn_activation_mult::mult(
    const bbts::ud_impl_t::tensor_params_t &params,
    const bbts::ud_impl_t::tensor_args_t &_in,
    bbts::ud_impl_t::tensor_args_t &_out) {

  // get the tensors as dense tensors
  auto &a = _in.get<0>().as<ffnn_dense_t>();
  auto &b = _in.get<1>().as<ffnn_dense_t>();
  auto &out = _out.get<0>().as<ffnn_dense_t>();

  // get the meta for the tensors
  auto &m_a = a.meta().m();
  auto &m_b = b.meta().m();
  auto &m_out = out.meta().m();

  // get the sizes
  uint32_t I = m_a.num_rows;
  uint32_t J = m_b.num_cols;
  uint32_t K = m_a.num_cols;

  // make sure the matrix size matches,
  // this is only present during the debug build
  assert(m_a.num_cols == m_b.num_rows);
  assert(m_b.has_bias);

  // get the ptrs
  float *outData = out.data();
  float *in1Data = a.data();
  float *in2Data = b.data();

  // set the new meta data
  m_out = {.num_rows = I,
           .num_cols = J,
           .row_idx = m_a.row_idx,
           .col_idx = m_b.col_idx,
           .has_bias = false,
           .num_aggregated = 1};

  // do the multiply
  float alpha = 1.0f;
  float beta = 0.0f;
  hipblasSgemm(params.cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, I, J, K, &alpha,
              in1Data, I, in2Data, K, &beta, outData, I);

  if (m_a.col_idx == 0 && m_b.row_idx == 0) {

    dim3 threadsPerBlock(8, 8);
    dim3 block_size((int)ceil((float)I / 8), (int)ceil((float)J / 8));

    ffnn_activation_mult_bias_add_kernel<<<block_size, threadsPerBlock, 0,
                                           params.stream>>>(b.bias(),
                                                            out.data(), I, J);
  }
}